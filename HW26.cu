#include "hip/hip_runtime.h"
// Name: Mason Bane
// nBody run on all available GPUs. 
// nvcc -use_fast_math HW26.cu -o temp -lglut -lm -lGLU -lGL

/*
 What to do:
 This is some robust N-body code with all the bells and whistles removed. 
 It automatically detects the number of available GPUs on the machine and runs using all of them.
 Rewrite the code using CUDA unified memory to make it simpler and use cudaMemPrefetchAsyn to make
 the memory movement faster.
*/

// Include files
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Defines
#define BLOCK_SIZE 128
#define PI 3.14159265359
#define DRAW_RATE 10

// This is to create a Lennard-Jones type function G/(r^p) - H(r^q). (p < q) p has to be less than q.
// In this code we will keep it a p = 2 and q = 4 problem. The diameter of a body is found using the general
// case so it will be more robust but in the code leaving it as a set 2, 4 problem make the coding much easier.
#define G 10.0f
#define H 10.0f
#define LJP  2.0
#define LJQ  4.0

#define DT 0.0001
#define RUN_TIME 1.0

// Globals
int N, OriginalN;
int NPerGPU; // Amount of vector on each GPU.
int NumberOfGpus;
float3 *P, *V, *F;
float *M; 
int *OffsetGPU; // Store offsets for each GPU
float GlobeRadius, Diameter, Radius;
float Damp;
dim3 BlockSize;
dim3 GridSize;
bool isFirstRun = true;

// Function prototypes
void cudaErrorCheck(const char *, int);
void drawPicture();
void setup();
__global__ void getForces(float3 *, float3 *, float3 *, float *, float, float, int, int, int);
__global__ void moveBodies(float3 *, float3 *, float3 *, float *, float, float, float, int, int, int);
void nBody();
int main(int, char**);

void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

void drawPicture()
{
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	
	glColor3d(1.0,1.0,0.5);
	
	for(int i=0; i<OriginalN; i++)
	{
		glPushMatrix();
		glTranslatef(P[i].x, P[i].y, P[i].z);
		glutSolidSphere(Radius,20,20);
		glPopMatrix();
	}
	
	glutSwapBuffers();
}

void setup()
{
	float randomAngle1, randomAngle2, randomRadius;
	float d, dx, dy, dz;
	int test;

	// Add at the start of setup()
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("Device name: %s\n", prop.name);
	printf("Compute capability: %d.%d\n", prop.major, prop.minor);
	printf("Unified addressing: %d\n", prop.unifiedAddressing);
	
	N = 301;
	
	hipGetDeviceCount(&NumberOfGpus);
	if(NumberOfGpus == 0)
	{
		printf("\n Dude, you don't even have a GPU. Sorry, you can't play with us. Call NVIDIA and buy a GPU — loser!\n");
		exit(0);
	}
	else
	{
		printf("\n You will be running on %d GPU(s)\n", NumberOfGpus);
	}
	
	OriginalN = N; // Save the original number of bodies so we can use it later for drawing

	// Using % to find how far off N is from prefectly dividing N. Then making sure there is enough blocks to cover this. 
	//pad the number of bodies to be even for each GPU
	if(N % NumberOfGpus != 0) //if there is a remainder
	{
		//add padding
		N += (NumberOfGpus - N % NumberOfGpus); //#Gpus - remainder = #bodies to add

		//should yield an even number of bodies for each GPU, worked with 101 on 2 and 3 GPUs doing the math by hand
	}

	//Now allocate the CPU memory to send to the GPUs
	NPerGPU = N/NumberOfGpus; //number of bodies per GPU

	OffsetGPU = (int *)malloc(NumberOfGpus*sizeof(int));
	for(int i = 0; i < NumberOfGpus; i++)
	{
		OffsetGPU[i] = i*NPerGPU;
	}


	BlockSize.x = 128;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	GridSize.x = (NPerGPU - 1)/BlockSize.x + 1; // This gives us the correct number of blocks.
	GridSize.y = 1;
	GridSize.z = 1;
	
	Damp = 0.5;
	
	// !! Important: Setting the number of bodies a little bigger if it is not even or you will 
	// get a core dump because you will be copying memory you do not own. This only needs to be
	// done for positions but I did it for all for completness incase the code gets used for a
	// more complicated force function.
	
	//int nn = NumberOfGpus*NPerGPU; // This will be N%NumberOfGpus bigger than N to keep use in bounds.
	
    // Allocate unified memory - accessible from both CPU and GPU
    hipMallocManaged(&M, N*sizeof(float));
    cudaErrorCheck(__FILE__, __LINE__);
    hipMallocManaged(&P, N*sizeof(float3));
    cudaErrorCheck(__FILE__, __LINE__);
    hipMallocManaged(&V, N*sizeof(float3));
    cudaErrorCheck(__FILE__, __LINE__);
    hipMallocManaged(&F, N*sizeof(float3));
    cudaErrorCheck(__FILE__, __LINE__);
	
    	
	Diameter = pow(H/G, 1.0/(LJQ - LJP)); // This is the value where the force is zero for the L-J type force.
	Radius = Diameter/2.0;
	
	// Using the radius of a body and a 68% packing ratio to find the radius of a global sphere that should hold all the bodies.
	// Then we double this radius just so we can get all the bodies setup with no problems. 
	float totalVolume = float(N)*(4.0/3.0)*PI*Radius*Radius*Radius;
	totalVolume /= 0.68;
	float totalRadius = pow(3.0*totalVolume/(4.0*PI), 1.0/3.0);
	GlobeRadius = 2.0*totalRadius;
	
	// Randomly setting these bodies in the glaobal sphere and setting the initial velosity, inotial force, and mass.
	for(int i = 0; i < N; i++)
	{
		test = 0;
		while(test == 0)
		{
			// Get random position.
			randomAngle1 = ((float)rand()/(float)RAND_MAX)*2.0*PI;
			randomAngle2 = ((float)rand()/(float)RAND_MAX)*PI;
			randomRadius = ((float)rand()/(float)RAND_MAX)*GlobeRadius;
			P[i].x = randomRadius*cos(randomAngle1)*sin(randomAngle2);
			P[i].y = randomRadius*sin(randomAngle1)*sin(randomAngle2);
			P[i].z = randomRadius*cos(randomAngle2);
			
			// Making sure the body centers are at least a diameter apart.
			// If they are not throw these positions away and try again.
			test = 1;
			for(int j = 0; j < i; j++)
			{
				dx = P[i].x-P[j].x;
				dy = P[i].y-P[j].y;
				dz = P[i].z-P[j].z;
				d = sqrt(dx*dx + dy*dy + dz*dz);
				if(d < Diameter)
				{
					test = 0;
					break;
				}
			}
		}
	
		V[i].x = 0.0;
		V[i].y = 0.0;
		V[i].z = 0.0;
		
		F[i].x = 0.0;
		F[i].y = 0.0;
		F[i].z = 0.0;
		
		M[i] = 1.0;


	}

	//make sure the last body doesn't mess with other people, everything should be 0 but mass
	for(int i = OriginalN; i < N; i++)
	{			
		P[i].x = 10.0e6f; //now its so far away it won't mess with anyone

		//Extra bodies are now skippe din the force calculation, along with that if it moves (which it shouldn't ever)
		//it won't mess with anyone else because it is so far away. It's just letting us use the same code for all bodies.

	}

		
	printf("\n Setup finished.\n");
}

__global__ void getForces(float3 *p, float3 *v, float3 *f, float *m, float g, float h, int myN, int totalN, int offset)
{
    float dx, dy, dz, d, d2;
    float force_mag;
    
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int globalIdx = i + offset;
    
    if(i < myN)  // Check against local bounds (starts at 0, ends at myN-1)
    {
        f[i].x = 0.0f;
        f[i].y = 0.0f;
        f[i].z = 0.0f;
        
        for(int j = 0; j < totalN; j++)
        {
            if(globalIdx != j)  // Use global index for comparison of everyone
            {
                dx = p[j].x-p[globalIdx].x;  //compare my global to every one else's global
                dy = p[j].y-p[globalIdx].y;
                dz = p[j].z-p[globalIdx].z;
                d2 = dx*dx + dy*dy + dz*dz;
                d = sqrt(d2);
                
				//calculate based on global idx
                force_mag = (g*m[globalIdx]*m[j])/(d2) - (h*m[globalIdx]*m[j])/(d2*d2);

                f[i].x += force_mag*dx/d;//store the forces of my half in local
                f[i].y += force_mag*dy/d;
                f[i].z += force_mag*dz/d;
            }
        }
    }
}

__global__ void moveBodies(float3 *p, float3 *v, float3 *f, float *m, float damp, float dt, float t, int n, int totalN, int offset)
{    
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int globalIdx = i + offset;
    
    if(i < n)
    {
        if(t == 0.0f)
        {
			//update local vels by using local forces and global mass (divided by 2 bc these are special)
            v[i].x += ((f[i].x-damp*v[i].x)/m[globalIdx])*dt/2.0f;
            v[i].y += ((f[i].y-damp*v[i].y)/m[globalIdx])*dt/2.0f;
            v[i].z += ((f[i].z-damp*v[i].z)/m[globalIdx])*dt/2.0f;
        }
        else
        {
			//update local vels by using local forces and global mass
            v[i].x += ((f[i].x-damp*v[i].x)/m[globalIdx])*dt;
            v[i].y += ((f[i].y-damp*v[i].y)/m[globalIdx])*dt;
            v[i].z += ((f[i].z-damp*v[i].z)/m[globalIdx])*dt;
        }

		//Update my global pos
        p[globalIdx].x += v[i].x*dt;  
        p[globalIdx].y += v[i].y*dt;
        p[globalIdx].z += v[i].z*dt;
    }
}

void nBody()
{
	int    drawCount = 0; 
	float  t = 0.0;
	float dt = 0.0001;
	
	
	if(isFirstRun) //added a global so this only prints once
	{
		if(N > OriginalN)
		{
			printf("\n Simulation is *technically* using %d bodies due to padding %d bodies with 0.\n", N, N-OriginalN);
			printf(" Only %d bodies will be drawn and calculated.\n", OriginalN);
			printf(" And if something weird does happen, that guy is really far away.\n");
		}
		else
		{
			printf("\n Simulation is running with %d bodies.\n", N);
		}
	}

	
	while(t < RUN_TIME)
	{
		// Adjusting bodies
        for(int i = 0; i < NumberOfGpus; i++)
        {
            hipSetDevice(i);
            
            // Prefetch tha stuff... If the GPU can figure out where to go it can figure out what it needs, right?
			hipMemPrefetchAsync(P, N*sizeof(float3), i);  // All p
			hipMemPrefetchAsync(V + OffsetGPU[i], NPerGPU*sizeof(float3), i);  // My v
			hipMemPrefetchAsync(F + OffsetGPU[i], NPerGPU*sizeof(float3), i);  // My f
			hipMemPrefetchAsync(M, N*sizeof(float), i);  // All m
            
            // Launch kernels using the unified memory and stored offsets
			getForces<<<GridSize,BlockSize>>>(P, V + OffsetGPU[i], F + OffsetGPU[i], M, G, H, NPerGPU, OriginalN, OffsetGPU[i]);
			cudaErrorCheck(__FILE__, __LINE__);
			moveBodies<<<GridSize,BlockSize>>>(P, V + OffsetGPU[i], F + OffsetGPU[i], M, Damp, dt, t, NPerGPU, N, OffsetGPU[i]);
			cudaErrorCheck(__FILE__, __LINE__);

			//Yep... that guy is not moving
			//printf("Padded body is at %f %f %f\n", P[N-1].x, P[N-1].y, P[N-1].z);
        }
		
		// Syncing CPU with GPUs.
		for(int i = 0; i < NumberOfGpus; i++)
    	{
			hipSetDevice(i);
			hipDeviceSynchronize();
			cudaErrorCheck(__FILE__, __LINE__);
		}
		

		if(drawCount == DRAW_RATE) 
		{	
			drawPicture();
			drawCount = 0;
		}
		
		t += dt;
		drawCount++;
	}

	isFirstRun = false; 
}

int main(int argc, char** argv)
{
	setup();
	
	int XWindowSize = 1000;
	int YWindowSize = 1000;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("Nbody");
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	glutDisplayFunc(drawPicture);
	glutIdleFunc(nBody);
	
	float3 eye = {0.0f, 0.0f, 2.0f*GlobeRadius};
	float near = 0.2;
	float far = 5.0*GlobeRadius;
	
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, near, far);
	glMatrixMode(GL_MODELVIEW);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	gluLookAt(eye.x, eye.y, eye.z, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
	
	glutMainLoop();

	free(OffsetGPU);

    // Free unified memory
    hipFree(P);
    hipFree(V);
    hipFree(F);
    hipFree(M);

	return 0;
}

