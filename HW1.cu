// Name:
// nvcc HW1.cu -o temp
/*
 What to do:
 1. Understand every line of code and be able to explain it in class.
 2. Compile, run, and play around with the code.
*/

// Include files

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

// Defines
#define N 1000 // Length of the vector

// Global variables
float *A_CPU, *B_CPU, *C_CPU; 
float Tolerance = 0.00000001;

// Function prototypes
void allocateMemory();
void innitialize();
void addVectorsCPU(float*, float*, float*, int);
int  check(float*, int);
long elaspedTime(struct timeval, struct timeval);
void cleanUp();

//Allocating the memory we will be using.
void allocateMemory()
{	
	// Host "CPU" memory.				
	A_CPU = (float*)malloc(N*sizeof(float));
	B_CPU = (float*)malloc(N*sizeof(float));
	C_CPU = (float*)malloc(N*sizeof(float));
} //allacotase N number of 32 bit floats (allocate N floats)

//Loading values into the vectors that we will add.
void innitialize()
{
	for(int i = 0; i < N; i++)
	{		
		A_CPU[i] = (float)i;	
		B_CPU[i] = (float)(2*i);
	}
}//goes through the array and sets vals for each element

//Adding vectors a and b then stores result in vector c.
void addVectorsCPU(float *a, float *b, float *c, int n)
{
	for(int id = 0; id < n; id++)
	{ 
		c[id] = a[id] + b[id];
	}
} // adds the two arrays and stores them in C

// Checking to see if anything went wrong in the vector addition.
int check(float *c, int n)
{
	int id;
	double sum = 0.0;
	double m = n-1; // Needed the -1 because we start at 0.
	
	for(id = 0; id < n; id++)
	{ 
		sum += c[id]; //sum up all elements in C
	}
	
	if(abs(sum - 3.0*(m*(m+1))/2.0) < Tolerance) //No clue what this calculation is exactly but it checks to make sure it within a tolarnce bc floats are weird
	{
		return(1); //good
	}
	else 
	{
		return(0); //bad
	}
}

// Calculating elasped time.
long elaspedTime(struct timeval start, struct timeval end)
{
	// tv_sec = number of seconds past the Unix epoch 01/01/1970
	// tv_usec = number of microseconds past the current second.
	
	long startTime = start.tv_sec * 1000000 + start.tv_usec; // In microseconds.
	long endTime = end.tv_sec * 1000000 + end.tv_usec; // In microseconds

	// Returning the total time elasped in microseconds
	return endTime - startTime;
}//when did I start - when did I end = time elapsed

//Cleaning up memory after we are finished.
void CleanUp()
{
	// Freeing host "CPU" memory.
	free(A_CPU); 
	free(B_CPU); 
	free(C_CPU);
}// EZ

int main()
{
	timeval start, end;
	
	// Allocating the memory you will need.
	allocateMemory();
	
	// Putting values in the vectors.
	innitialize();

	// Starting the timer.	
	gettimeofday(&start, NULL);

	// Add the two vectors.
	addVectorsCPU(A_CPU, B_CPU ,C_CPU, N);

	// Stopping the timer.
	gettimeofday(&end, NULL);
	
	// Checking to see if all went correctly.
	if(check(C_CPU, N) == 0)
	{
		printf("\n\n Something went wrong in the vector addition\n");
	}
	else
	{
		printf("\n\n You added the two vectors correctly on the CPU");
		printf("\n The time it took was %ld microseconds", elaspedTime(start, end));
	}
	
	// Your done so cleanup your room.	
	CleanUp();	
	
	// Making sure it flushes out anything in the print buffer.
	printf("\n");
	
	return(0);
}//Calls functions and prints information based on what happened

