#include "hip/hip_runtime.h"
// Name: Mason Bane
// nBody run on all available GPUs. 
// nvcc HW25.cu -o temp -lglut -lm -lGLU -lGL

/*
 What to do:
 This is some robust N-body code with all the bells and whistles removed. 
 It runs on two GPUs and two GPUs only. Rewrite it so it automatically detects the number of 
 available GPUs on the machine and runs using all of them.
*/

// Include files
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Defines
#define BLOCK_SIZE 128
#define PI 3.14159265359
#define DRAW_RATE 10

// This is to create a Lennard-Jones type function G/(r^p) - H(r^q). (p < q) p has to be less than q.
// In this code we will keep it a p = 2 and q = 4 problem. The diameter of a body is found using the general
// case so it will be more robust but in the code leaving it as a set 2, 4 problem make the coding much easier.
#define G 10.0f
#define H 10.0f
#define LJP  2.0
#define LJQ  4.0

#define DT 0.0001
#define RUN_TIME 1.0

// Globals
int N;
int myN; //number of bodies per GPU
int NumberOfGpus;
float3 *P, *V, *F;
float *M; 

//GPU variables
float3 **PGPU, **VGPU, **FGPU; //double pointers for an array of pointers to the GPU memory
float **MGPU; //again....
int *NGPU; //pointer for # of bodies on each GPU
int *OffsetGPU; //pointer for offset of each GPU

float GlobeRadius, Diameter, Radius;
float Damp;
dim3 BlockSize;
dim3 GridSize;

// Function prototypes
void cudaErrorCheck(const char *, int);
void drawPicture();
void setup();
__global__ void getForces(float3 *, float3 *, float3 *, float *, float, float, int, int, int);
__global__ void moveBodies(float3 *, float3 *, float3 *, float *, float, float, float, int, int);
void nBody();
int main(int, char**);

void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

void drawPicture()
{
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	hipSetDevice(0);
	//copy the positions from the first GPU to the CPU for drawing (should have all updated positions)
	hipMemcpyAsync(P, PGPU[0], N*sizeof(float3), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);

	//random color for the bodies
	float randValue = ((float)rand()/(float)RAND_MAX);

	
	
	
	for(int i=0; i<N; i++)
	{

		float r = ((float)rand()/(float)RAND_MAX);
        float g = ((float)rand()/(float)RAND_MAX);
        float b = ((float)rand()/(float)RAND_MAX);

		glColor3d(r, g, b);
		
		glPushMatrix();
		glTranslatef(P[i].x, P[i].y, P[i].z);
		glutSolidSphere(Radius,20,20);
		glPopMatrix();
	}
	
	glutSwapBuffers();
}

void setup()
{
	float randomAngle1, randomAngle2, randomRadius;
	float d, dx, dy, dz;
	int test;
	
	N = 501;
	
	hipGetDeviceCount(&NumberOfGpus);
	if(NumberOfGpus < 1 && NumberOfGpus >= 0)
	{
		printf("\n\n\nWhy are you trying to run CUDA Code with no GPU?\n");
		printf("You must be either broke or dumb, but one thing's for certain, you're definitely a LOSER.\n");
		printf("Go get a job or get a really rich partner, get your life together, and buy a GPU.......\n");
		printf("Until then, I dont want to see your ugly mug running this code again... I'm outta here\n\n");
		exit(0);

	}
	else if(NumberOfGpus >= 1)
	{
		printf("\n\n Running with %d GPUs\n", NumberOfGpus);

		//allocate arrays to hold pointers to GPU memory
		PGPU = (float3 **)malloc(NumberOfGpus*sizeof(float3*));
		VGPU = (float3 **)malloc(NumberOfGpus*sizeof(float3*));
		FGPU = (float3 **)malloc(NumberOfGpus*sizeof(float3*));

		MGPU = (float **)malloc(NumberOfGpus*sizeof(float*));

		NGPU = (int *)malloc(NumberOfGpus*sizeof(int));
		OffsetGPU = (int *)malloc(NumberOfGpus*sizeof(int));

		//pad the number of bodies to be even for each GPU
		if(N % NumberOfGpus != 0) //if there is a remainder
		{
			//add padding
			N += (NumberOfGpus - N % NumberOfGpus); //#Gpus - remainder = #bodies to add

			//should yield an even number of bodies for each GPU, worked with 101 on 2 and 3 GPUs doing the math by hand
		}

		//Now allocate the CPU memory to send to the GPUs
		myN = N/NumberOfGpus; //number of bodies per GPU

		for(int i=0; i<NumberOfGpus; i++)
		{
			PGPU[i] = (float3 *)malloc(myN*sizeof(float3));
			VGPU[i] = (float3 *)malloc(myN*sizeof(float3));
			FGPU[i] = (float3 *)malloc(myN*sizeof(float3));

			MGPU[i] = (float *)malloc(myN*sizeof(float));
			NGPU[i] = myN;
			OffsetGPU[i] = i*myN; //pretty cool huh?
		}

		//Now each GPU should be able to also have the same amt of blocks and threads

		BlockSize.x = BLOCK_SIZE;
		BlockSize.y = 1;
		BlockSize.z = 1;

		GridSize.x = (myN + BlockSize.x - 1)/BlockSize.x; //this is the number of blocks needed to cover all the bodies
		GridSize.y = 1;
		GridSize.z = 1;

		//allocate memory on the GPU(s)
		for(int i=0; i<NumberOfGpus; i++)
		{
			hipSetDevice(i);
			hipMalloc((void**)&PGPU[i], N*sizeof(float3)); //all
			cudaErrorCheck(__FILE__, __LINE__);
			hipMalloc((void**)&VGPU[i], myN*sizeof(float3)); //mine
			cudaErrorCheck(__FILE__, __LINE__);
			hipMalloc((void**)&FGPU[i], myN*sizeof(float3)); //mine
			cudaErrorCheck(__FILE__, __LINE__);
			hipMalloc((void**)&MGPU[i], N*sizeof(float)); //all
			cudaErrorCheck(__FILE__, __LINE__);
		}		
	}
	else
	{
		printf("\n\n\n You have either a negative or uncountable number of GPUs....\n");
		printf("How the he(ll)(ck if your an employer looking at this) did you even hit this block?.... Enjoy putting out the fire\n\n");
		exit(0);
	}

	//Now we can finally let the CPU code do its thing......
	
	Damp = 0.5;
	
	M = (float*)malloc(N*sizeof(float));
	P = (float3*)malloc(N*sizeof(float3));
	V = (float3*)malloc(N*sizeof(float3));
	F = (float3*)malloc(N*sizeof(float3));
	
	// !! Important: Setting the number of bodies a little bigger if it is not even or you will 
	// get a core dump because you will be copying memory you do not own. This only needs to be
	// done for positions but I did it for all for completness incase the code gets used for a
	// more complicated force function.
    	
	Diameter = pow(H/G, 1.0/(LJQ - LJP)); // This is the value where the force is zero for the L-J type force.
	Radius = Diameter/2.0;
	
	// Using the radius of a body and a 68% packing ratio to find the radius of a global sphere that should hold all the bodies.
	// Then we double this radius just so we can get all the bodies setup with no problems. 
	float totalVolume = float(N)*(4.0/3.0)*PI*Radius*Radius*Radius;
	totalVolume /= 0.68;
	float totalRadius = pow(3.0*totalVolume/(4.0*PI), 1.0/3.0);
	GlobeRadius = 2.0*totalRadius;
	
	// Randomly setting these bodies in the glaobal sphere and setting the initial velosity, inotial force, and mass.
	for(int i = 0; i < N; i++)
	{
		test = 0;
		while(test == 0)
		{
			// Get random position.
			randomAngle1 = ((float)rand()/(float)RAND_MAX)*2.0*PI;
			randomAngle2 = ((float)rand()/(float)RAND_MAX)*PI;
			randomRadius = ((float)rand()/(float)RAND_MAX)*GlobeRadius;
			P[i].x = randomRadius*cos(randomAngle1)*sin(randomAngle2);
			P[i].y = randomRadius*sin(randomAngle1)*sin(randomAngle2);
			P[i].z = randomRadius*cos(randomAngle2);
			
			// Making sure the bodies' centers are at least a diameter apart.
			// If they are not throw these positions away and try again.
			test = 1;
			for(int j = 0; j < i; j++)
			{
				dx = P[i].x-P[j].x;
				dy = P[i].y-P[j].y;
				dz = P[i].z-P[j].z;
				d = sqrt(dx*dx + dy*dy + dz*dz);
				if(d < Diameter)
				{
					test = 0;
					break;
				}
			}
		}
	
		V[i].x = 0.0;
		V[i].y = 0.0;
		V[i].z = 0.0;
		
		F[i].x = 0.0;
		F[i].y = 0.0;
		F[i].z = 0.0;
		
		M[i] = 1.0;
	}

	//Alright now that the CPUs done wiith all that we can send the data to the GPUs.

	for(int i = 0; i < NumberOfGpus; i++)
	{
		hipSetDevice(i);
		hipMemcpyAsync(PGPU[i], P, N*sizeof(float3), hipMemcpyHostToDevice); //all
		cudaErrorCheck(__FILE__, __LINE__);
		hipMemcpyAsync(VGPU[i], V, myN*sizeof(float3), hipMemcpyHostToDevice); //mine
		cudaErrorCheck(__FILE__, __LINE__);
		hipMemcpyAsync(FGPU[i], F, myN*sizeof(float3), hipMemcpyHostToDevice); //mine
		cudaErrorCheck(__FILE__, __LINE__);
		hipMemcpyAsync(MGPU[i], M, N*sizeof(float), hipMemcpyHostToDevice); //all
		cudaErrorCheck(__FILE__, __LINE__);
	}
		
	printf("\n Setup finished.\n");
}

__global__ void getForces(float3 *p, float3 *v, float3 *f, float *m, float g, float h, int myN, int totalN, int offset)
{
    float dx, dy, dz, d, d2;
    float force_mag;
    
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int globalIdx = i + offset;
    
    if(i < myN)  // Check against local bounds (starts at 0, ends at myN-1)
    {
        f[i].x = 0.0f;
        f[i].y = 0.0f;
        f[i].z = 0.0f;
        
        for(int j = 0; j < totalN; j++)
        {
            if(globalIdx != j)  // Use global index for comparison of everyone
            {
                dx = p[j].x-p[globalIdx].x;  //compare my global to every one else's global
                dy = p[j].y-p[globalIdx].y;
                dz = p[j].z-p[globalIdx].z;
                d2 = dx*dx + dy*dy + dz*dz;
                d = sqrt(d2);
                
				//calculate based on global idx
                force_mag = (g*m[globalIdx]*m[j])/(d2) - (h*m[globalIdx]*m[j])/(d2*d2);

                f[i].x += force_mag*dx/d;//store the forces of my half in local
                f[i].y += force_mag*dy/d;
                f[i].z += force_mag*dz/d;
            }
        }
    }
}

__global__ void moveBodies(float3 *p, float3 *v, float3 *f, float *m, float damp, float dt, float t, int n, int offset)
{    
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int globalIdx = i + offset;
    
    if(i < n)
    {
        if(t == 0.0f)
        {
			//update local vels by using local forces and global mass (divided by 2 bc these are special)
            v[i].x += ((f[i].x-damp*v[i].x)/m[globalIdx])*dt/2.0f;
            v[i].y += ((f[i].y-damp*v[i].y)/m[globalIdx])*dt/2.0f;
            v[i].z += ((f[i].z-damp*v[i].z)/m[globalIdx])*dt/2.0f;
        }
        else
        {
			//update local vels by using local forces and global mass
            v[i].x += ((f[i].x-damp*v[i].x)/m[globalIdx])*dt;
            v[i].y += ((f[i].y-damp*v[i].y)/m[globalIdx])*dt;
            v[i].z += ((f[i].z-damp*v[i].z)/m[globalIdx])*dt;
        }

		//Update my global pos
        p[globalIdx].x += v[i].x*dt;  
        p[globalIdx].y += v[i].y*dt;
        p[globalIdx].z += v[i].z*dt;
    }
}


void nBody()
{
	int    drawCount = 0; 
	float  t = 0.0;
	float dt = 0.0001;

	while(t < RUN_TIME)
	{
		for(int i = 0; i < NumberOfGpus; i++)//for each device
		{
			//set the device and run its kernel
			hipSetDevice(i);
			getForces<<<GridSize,BlockSize>>>(PGPU[i], VGPU[i], FGPU[i], MGPU[i], G, H, myN, N, OffsetGPU[i]);
			cudaErrorCheck(__FILE__, __LINE__);
			moveBodies<<<GridSize,BlockSize>>>(PGPU[i], VGPU[i], FGPU[i], MGPU[i], Damp, dt, t, myN, OffsetGPU[i]);
			cudaErrorCheck(__FILE__, __LINE__);
		}

		//sync all devices to make sure they are done before we copy memory between them
		for(int i = 0; i < NumberOfGpus; i++)//for each device
		{
			hipSetDevice(i);
			hipDeviceSynchronize();
			cudaErrorCheck(__FILE__, __LINE__);
		}

		if(NumberOfGpus > 1) //if we have 1 GPU, the device has everything updated already
		{
			for(int src = 0; src < NumberOfGpus; src++)//for each device (the source)
			{
				for(int dst = 0; dst < NumberOfGpus; dst++)//copy my data to everyone else(destination)
				{
					if(src != dst) //if not the same device
					{
						hipSetDevice(dst);
						hipMemcpyAsync(PGPU[dst] + OffsetGPU[src],PGPU[src] + OffsetGPU[src], myN*sizeof(float3), hipMemcpyDeviceToDevice);
						cudaErrorCheck(__FILE__, __LINE__);
					}
				}
			}

			//make sure we've gone around the circle and shared our data with eachother
			// GPUs anonymous

			for(int i = 0; i < NumberOfGpus; i++)//for each device
			{
				hipSetDevice(i);
				hipDeviceSynchronize();
				cudaErrorCheck(__FILE__, __LINE__);
			}

		}


		if(drawCount == DRAW_RATE) 
		{	
			drawPicture();
			drawCount = 0;
		}
		
		t += dt;
		drawCount++;
	}
}

int main(int argc, char** argv)
{
	setup();
	
	int XWindowSize = 1000;
	int YWindowSize = 1000;
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("Nbody N GPUs");
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	glutDisplayFunc(drawPicture);
	glutIdleFunc(nBody);
	
	float3 eye = {0.0f, 0.0f, 2.0f*GlobeRadius};
	float near = 0.2;
	float far = 5.0*GlobeRadius;
	
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, near, far);
	glMatrixMode(GL_MODELVIEW);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	gluLookAt(eye.x, eye.y, eye.z, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
	
	glutMainLoop();
	return 0;
}

