// Name: Mason
// CPU random walk. 
// nvcc HW28.cu -o temp

/*
 What to do:
 This is some code that runs a random walk for 10000 steps.
 Use cudaRand and run 10 of these runs at once with diferent seeds on the GPU.
 Print out all 10 final positions.
*/

// Include files

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h> //for random number generation


// Defines
#define NUM_WALKS 10 // Number of walks to perform
#define N 10'000 // Number of steps per walk
#define BLOCK_SIZE 256 // Number of threads per block


// Globals
dim3 BlockSize;
dim3 GridSize;
int *distance, *distanceGPU; // Array to store distances for each walk


// Function prototypes
int walk(int);
__global__ void walkGPU(); // Device function to generate random step
bool setup();


bool setup()
{
	BlockSize.x = BLOCK_SIZE;
	BlockSize.y = 1;
	BlockSize.z = 1;

	GridSize.x = (NUM_WALKS + BlockSize.x - 1) / BlockSize.x; // Number of blocks needed( for 10 walks and 256 threads, we need 1 block) (10+256-1)/256 = 265/256 = 1
	GridSize.y = 1;
	GridSize.z = 1;

	distance = (int*)malloc(NUM_WALKS * sizeof(int)); // Allocate memory for distance on host

	hipMalloc(&distanceGPU, NUM_WALKS * sizeof(int)); // Allocate memory for distance on device

	//no need for copies until we are done with the kernel

	return true;
}

__global__ void walkGPU(int *distanceGPU, unsigned int seed)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int position;

	if (id < NUM_WALKS)
	{
		hiprandState state; // random state for each thread, the state is a data structure that CUDA uses for its RNG algorithms
		hiprand_init(seed, id, 0, &state); // Initialize the random state, args (seed, thread id, sequence number[allows multiple RNG streams
		position = 0; // Reset position for each walk

		for (int i = 0; i < N; i++)
		{
			// Random number -1 or 1
			int step = hiprand(&state) % 2 * 2 - 1; // number%2 [0, 1] then multiply by 2 [0, 2] and subtract 1 [-1, 1]

			// Update position
			position += step;
		}

		distanceGPU[id] = position; // Store the final position in global memory
	}
}


int main(int argc, char** argv)
{


	if(!setup()) return -1;

	//Do the GPU walks, only need 1 kenrel call for all walks
	walkGPU<<<GridSize, BlockSize>>>(distanceGPU, time(NULL));

	// Wait for the kernel to finish
	hipDeviceSynchronize(); 

	// Copy the results back to host
	hipMemcpy(distance, distanceGPU, NUM_WALKS * sizeof(int), hipMemcpyDeviceToHost);

	//print results
	printf("Final positions after %d walks:\n", NUM_WALKS);
	for (int i = 0; i < NUM_WALKS; i++)
	{
		printf("Walk %d Final Position: %d\n", i + 1, distance[i]);
	}


	//clean up
	free(distance); 
	hipFree(distanceGPU);
	
	return 0;
}

